#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "readFloatFromInt16.h"
#include "dft.h"
#include "writeFloat.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stopwatch.h"

/*
dim3 gridDim x, y
dim3 blockDim x, y, z
dim3 blockIdx
dim3 threadIdx
*/
__global__ void vector_dft(float* src, size_t len, float* re, float* im) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return;
  }
  float re1 = 0.0f;
  float im1 = 0.0f;
  float dth = (2.0f * (float)M_PI) * i / len;
  float th = 0.0f;
  for (int j = 0; j < len; j++) {
    float d = src[j];
    re1 += d * cosf(th);
    im1 -= d * sinf(th);
    th += dth;
  }
  re[i] = re1;
  im[i] = im1;
}

int check(int nthread, float* src, size_t len, float* re, float* im) {
  clock_t nowc = clock();
  stopwatch_start();

  // Allocate memory
  float* d_src;
  float* d_re;
  float* d_im;
  hipMalloc((void**)&d_src, sizeof(float) * len);
  hipMalloc((void**)&d_re, sizeof(float) * len);
  hipMalloc((void**)&d_im, sizeof(float) * len);

  hipMemcpy(d_src, src, sizeof(float) * len, hipMemcpyHostToDevice);

  int nblock = len / nthread + (len % nthread == 0 ? 0 : 1);
  vector_dft<<<nblock, nthread>>>(d_src, len, d_re, d_im);
   
  hipMemcpy(re, d_re, sizeof(float) * len, hipMemcpyDeviceToHost);
  hipMemcpy(im, d_im, sizeof(float) * len, hipMemcpyDeviceToHost);

  // Deallocate device memory
  hipFree(d_src);
  hipFree(d_re);
  hipFree(d_im);

  float dt =   stopwatch_stop();
  double dtc = (double)(clock() - nowc) / CLOCKS_PER_SEC;

  printf("%d,", nthread);
  printf("%d,", nblock);
  printf("%f,", dt);
  printf("%f\n", dtc);
  return 0;
}

int main(void) {
  int benchmark = 0;

  const char* fn = "sekaideichiban.wav-r.i16.bin";
  const char* fnre = "sekaideichiban.wav-re.f32.bin";
  const char* fnim = "sekaideichiban.wav-im.f32.bin";
  
  size_t len = 0;
  float* src = readFloatFromInt16(fn, &len);
  if (!src) {
    return 1;
  }
  printf("len: %zu\n", len);

  float* re = (float*)malloc(len * sizeof(float));
  float* im = (float*)malloc(len * sizeof(float));

  for (int i = 0; i < len; i++) {
    re[i] = i;
  }

  printf("nthread,nblock,time,cputime\n");
  if (benchmark) {
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    // Maximum number of threads per block = 1024
    for (int i = 1024; i >= 1; i >>= 1) {
      check(i, src, len, re, im);
    }
  } else {
    check(1024, src, len, re, im); // for RXT3060Ti
  }
  /*
  for (int i = 0; i < len; i++) {
    printf("%d %f\n", i, re[i]);
  }
  */
  writeFloat(fnre, re, len);
  writeFloat(fnim, im, len);

  free(re);
  free(im);
  free(src);

  return 0;
}
