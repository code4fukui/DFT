#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "readFloat.h"
#include "writeInt16.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stopwatch.h"
#include "float2short.h"

__global__ void vector_idft(float* re, float* im, size_t len, float* dst) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return;
  }
  float re1 = 0.0f;
  for (int j = 0; j < len; j++) {
    float th = (2.0f * (float)M_PI) * i * j / len;
    re1 += re[j] * cosf(th) - im[j] * sinf(th);
  }
  dst[i] = re1 / len;
}

int check(int nthread, float* re, float* im, size_t len, float* dst) {
  clock_t nowc = clock();
  stopwatch_start();

  // Allocate memory
  float* d_re;
  float* d_im;
  float* d_dst;
  hipMalloc((void**)&d_re, sizeof(float) * len);
  hipMalloc((void**)&d_im, sizeof(float) * len);
  hipMalloc((void**)&d_dst, sizeof(float) * len);

  hipMemcpy(d_re, re, sizeof(float) * len, hipMemcpyHostToDevice);
  hipMemcpy(d_im, im, sizeof(float) * len, hipMemcpyHostToDevice);

  int nblock = len / nthread + (len % nthread == 0 ? 0 : 1);
  vector_idft<<<nblock, nthread>>>(d_re, d_im, len, d_dst);
   
  hipMemcpy(dst, d_dst, sizeof(float) * len, hipMemcpyDeviceToHost);

  // Deallocate device memory
  hipFree(d_re);
  hipFree(d_im);
  hipFree(d_dst);

  float dt = stopwatch_stop();
  double dtc = (double)(clock() - nowc) / CLOCKS_PER_SEC;

  printf("%d,", nthread);
  printf("%d,", nblock);
  printf("%f,", dt);
  printf("%f\n", dtc);
  return 0;
}

int main(void) {
  int benchmark = 0;
  printf("benchmark mode: %d\n", benchmark);
  
  const char* fnre = "sekaideichiban.wav-re.f32.bin";
  const char* fnim = "sekaideichiban.wav-im.f32.bin";
  const char* fn2 = "sekaideichiban.wav-r-idft.i16.bin";
  
  size_t len = 0;
  float* re = readFloat(fnre, &len);
  float* im = readFloat(fnim, &len);
  if (!re || !im) {
    printf("can't read src");
    return 1;
  }
  printf("len: %zu\n", len);

  float* dst = (float*)malloc(len * sizeof(float));

  printf("nthread,nblock,time,cputime\n");
  if (benchmark) {
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    // Maximum number of threads per block = 1024
    for (int i = 1024; i >= 1; i >>= 1) {
      check(i, re, im, len, dst);
    }
  } else {
    check(1024, re, im, len, dst);
  }
  /*
  for (int i = 0; i < len; i++) {
    printf("%d %f\n", i, re[i]);
  }
  */
  writeInt16(fn2, float2short(dst, len), len);

  free(re);
  free(im);
  free(dst);

  return 0;
}
