#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "readDoubleFromInt16.h"
#include "dft.h"
#include "writeDouble.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stopwatch.h"

/*
dim3 gridDim x, y
dim3 blockDim x, y, z
dim3 blockIdx
dim3 threadIdx
*/
__global__ void vector_dft(double* src, size_t len, double* re, double* im) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return;
  }
  double re1 = 0.0;
  double im1 = 0.0;
  for (int j = 0; j < len; j++) {
    double th = (2.0 * M_PI) * i * j / len;
    double d = src[j];
    re1 += d * cos(th);
    im1 -= d * sin(th);
  }
  re[i] = re1;
  im[i] = im1;
}

int check(int nthread, double* src, size_t len, double* re, double* im) {
  clock_t nowc = clock();
  stopwatch_start();

  // Allocate memory
  double* d_src;
  double* d_re;
  double* d_im;
  hipMalloc((void**)&d_src, sizeof(double) * len);
  hipMalloc((void**)&d_re, sizeof(double) * len);
  hipMalloc((void**)&d_im, sizeof(double) * len);

  hipMemcpy(d_src, src, sizeof(double) * len, hipMemcpyHostToDevice);

  int nblock = len / nthread + (len % nthread == 0 ? 0 : 1);
  vector_dft<<<nblock, nthread>>>(d_src, len, d_re, d_im);
   
  hipMemcpy(re, d_re, sizeof(double) * len, hipMemcpyDeviceToHost);
  hipMemcpy(im, d_im, sizeof(double) * len, hipMemcpyDeviceToHost);

  // Deallocate device memory
  hipFree(d_src);
  hipFree(d_re);
  hipFree(d_im);

  double dt = stopwatch_stop();
  double dtc = (double)(clock() - nowc) / CLOCKS_PER_SEC;

  printf("%d,", nthread);
  printf("%d,", nblock);
  printf("%f,", dt);
  printf("%f\n", dtc);
  return 0;
}

int main(void) {
  int benchmark = 0;

  const char* fn = "sekaideichiban.wav-r.i16.bin";
  const char* fnre = "sekaideichiban.wav-re.f64.bin";
  const char* fnim = "sekaideichiban.wav-im.f64.bin";
  
  size_t len = 0;
  double* src = readDoubleFromInt16(fn, &len);
  if (!src) {
    return 1;
  }
  printf("len: %zu\n", len);

  double* re = (double*)malloc(len * sizeof(double));
  double* im = (double*)malloc(len * sizeof(double));

  for (int i = 0; i < len; i++) {
    re[i] = i;
  }

  printf("nthread,nblock,time,cputime\n");
  if (benchmark) {
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    // Maximum number of threads per block = 1024
    for (int i = 1024; i >= 1; i >>= 1) {
      check(i, src, len, re, im);
    }
  } else {
    check(1024, src, len, re, im); // for RXT3060Ti
  }
  /*
  for (int i = 0; i < len; i++) {
    printf("%d %f\n", i, re[i]);
  }
  */
  writeDouble(fnre, re, len);
  writeDouble(fnim, im, len);

  free(re);
  free(im);
  free(src);

  return 0;
}
