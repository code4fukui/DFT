#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "readDouble.h"
#include "writeInt16.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stopwatch.h"
#include "double2short.h"

__global__ void vector_idft(double* re, double* im, size_t len, double* dst) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return;
  }
  double re1 = 0.0;
  for (int j = 0; j < len; j++) {
    double th = (2.0 * M_PI) * i * j / len;
    re1 += re[j] * cos(th) - im[j] * sin(th);
  }
  dst[i] = re1 / len;
}

int check(int nthread, double* re, double* im, size_t len, double* dst) {
  clock_t nowc = clock();
  stopwatch_start();

  // Allocate memory
  double* d_re;
  double* d_im;
  double* d_dst;
  hipMalloc((void**)&d_re, sizeof(double) * len);
  hipMalloc((void**)&d_im, sizeof(double) * len);
  hipMalloc((void**)&d_dst, sizeof(double) * len);

  hipMemcpy(d_re, re, sizeof(double) * len, hipMemcpyHostToDevice);
  hipMemcpy(d_im, im, sizeof(double) * len, hipMemcpyHostToDevice);

  int nblock = len / nthread + (len % nthread == 0 ? 0 : 1);
  vector_idft<<<nblock, nthread>>>(d_re, d_im, len, d_dst);
  
  hipMemcpy(dst, d_dst, sizeof(double) * len, hipMemcpyDeviceToHost);

  // Deallocate device memory
  hipFree(d_re);
  hipFree(d_im);
  hipFree(d_dst);

  double dt = stopwatch_stop();
  double dtc = (double)(clock() - nowc) / CLOCKS_PER_SEC;

  printf("%d,", nthread);
  //printf("%d,", nblock);
  printf("%f,", dt);
  printf("%f\n", dtc);
  return 0;
}

int main(void) {
  int benchmark = 0;
  printf("benchmark mode: %d\n", benchmark);
  
  const char* fnre = "sekaideichiban.wav-re.f64.bin";
  const char* fnim = "sekaideichiban.wav-im.f64.bin";
  const char* fn2 = "sekaideichiban.wav-r-idft.i16.bin";
  
  size_t len = 0;
  double* re = readDouble(fnre, &len);
  double* im = readDouble(fnim, &len);
  if (!re || !im) {
    printf("can't read src");
    return 1;
  }
  printf("len: %zu\n", len);

  double* dst = (double*)malloc(len * sizeof(double));

  printf("nthread,nblock,time,cputime\n");
  if (benchmark) {
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
    // Maximum number of threads per block = 1024
    for (int i = 1024; i >= 1; i >>= 1) {
      check(i, re, im, len, dst);
    }
  } else {
    check(1024, re, im, len, dst);
  }
  /*
  for (int i = 0; i < len; i++) {
    printf("%d %f\n", i, re[i]);
  }
  */
  writeInt16(fn2, double2short(dst, len), len);

  free(re);
  free(im);
  free(dst);

  return 0;
}
